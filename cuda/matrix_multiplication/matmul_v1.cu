
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <cstdlib>
#include <cmath>
#include <cblas.h>
#include <omp.h>
#include <random>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

#define ceil_div(a, b) ((a + b - 1) / b)


const int DSIZE = 8 * 1024;
const int BLOCK_SIZE = 1024;
const int GRID_SIZE = 1024;

// Most naive matmul kernel, using blocks for rows of A and threads for cols of B
__global__ void matmul(const float *A, const float *B, float *output, int ds){
  float accu;
  for (int row_id = blockIdx.x; row_id < ds; row_id += gridDim.x) {
    for (int col_id = threadIdx.x; col_id < ds; col_id += blockDim.x) {
      accu = 0.0;
      for (int i = 0; i < ds; i++) {
        accu += A[row_id * ds + i] * B[i * ds + col_id];
      }
      output[row_id * ds + col_id] = accu;
    }
  }
}

int main(){
  float *host_matrix_A = new float[DSIZE * DSIZE];
  float *host_matrix_B = new float[DSIZE * DSIZE];
  float *host_output = new float[DSIZE * DSIZE];
  float *answer = new float[DSIZE * DSIZE];
  float *device_matrix_A, *device_matrix_B, *device_output;
  printf("Filling in random data.\n");
  #pragma omp parallel
  {
    std::random_device rd;
    std::mt19937 gen(rd() + omp_get_thread_num()); // Seed with thread ID
    std::uniform_real_distribution<float> dis(-1.0f, 1.0f);

    #pragma omp for collapse(2)
    for (int i = 0; i < DSIZE; i++) {
      for (int j = 0; j < DSIZE; j++) {
        host_matrix_A[i * DSIZE + j] = dis(gen);
        host_matrix_B[i * DSIZE + j] = dis(gen);
      }
    }
  }

  printf("Computing answer on CPU.\n");
  cblas_sgemm(
    CblasRowMajor,
    CblasNoTrans,
    CblasNoTrans,
    DSIZE,
    DSIZE,
    DSIZE,
    1.0f,
    host_matrix_A,
    DSIZE,
    host_matrix_B,
    DSIZE,
    1.0,
    answer,
    DSIZE
  );



  hipMalloc(&device_matrix_A, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&device_matrix_B, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&device_output, DSIZE*DSIZE*sizeof(float));

  cudaCheckErrors("hipMalloc failure"); // error checking
  hipMemcpy(device_matrix_A, host_matrix_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_matrix_B, host_matrix_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  matmul<<<GRID_SIZE, BLOCK_SIZE>>>(device_matrix_A, device_matrix_B, device_output, DSIZE);

  hipEventRecord(stop);
  cudaCheckErrors("kernel launch failure");
  hipMemcpy(host_output, device_output, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time: %f ms\n", milliseconds);

  printf("Checking answers......");
  for(int i = 0; i < DSIZE; i++) {
    for(int j = 0; j < DSIZE; j++) {
      if (abs(host_output[i * DSIZE + j] - answer[i * DSIZE + j]) >= 1e-1) {
        printf("Incorrect result! GPU: %f, CPU: %f\n", host_output[i * DSIZE + j], answer[i * DSIZE + j]);
        return -1;
      }
    }
  }
  printf(" passed!\n");

  return 0;
}

