
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const int DSIZE = 256 * 1024 * 1024; // 1GB of data
const int block_size = 1024;  // CUDA maximum is 1024
// vector add kernel: C = A + B
__global__ void vadd(const float *A, const float *B, float *C, int ds){

  int idx = block_size * blockIdx.x + threadIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds)
    C[idx] = A[idx] + B[idx]; // do the vector (element) add here
}

int main(){

  float *h_A, *h_B, *h_C, *h_S, *d_A, *d_B, *d_C;
  h_A = new float[DSIZE];  // allocate space for vectors in host memory
  h_B = new float[DSIZE];
  h_C = new float[DSIZE];
  h_S = new float[DSIZE];
  for (int i = 0; i < DSIZE; i++){  // initialize vectors in host memory
    h_A[i] = rand()/(float)RAND_MAX;
    h_B[i] = rand()/(float)RAND_MAX;
    h_C[i] = 0;
    h_S[i] = h_A[i] + h_B[i];
  }
  hipMalloc(&d_A, DSIZE*sizeof(float));  // allocate device space for vector A
  hipMalloc(&d_B, DSIZE*sizeof(float));  // allocate device space for vector B
  hipMalloc(&d_C, DSIZE*sizeof(float));  // allocate device space for vector C
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy vector A to device:
  hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  // copy vector B to device:
  hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  vadd<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_B, d_C, DSIZE);
  hipEventRecord(stop);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector C from device to host:
  hipMemcpy(h_C, d_C, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  float memory_gbps = 2 * float(DSIZE) * 4 / 1024 / 1024 / 1024 / (milliseconds / 1000);
  printf("Time: %f ms, Memory Bandwidth: %f GB/s\n", milliseconds, memory_gbps);
  printf("Checking correctness...... ");
  for (int i = 0; i < DSIZE; i++){
    assert(h_S[i] == h_C[i] && "Incorrect result!");
  }
  printf("passed!\n");
  return 0;
}

